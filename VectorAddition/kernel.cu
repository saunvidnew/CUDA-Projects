﻿#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
__global__ void vecAdd(int *A, int *B, int *C, int n) {
	
	int i = threadIdx.x;

	if (i < n) {
		C[i] = A[i] + B[i];
	}
	
		

}

int main() {
	int *A, *B, *C;
	int n=4;
	hipMallocManaged(&A, n * sizeof(int));
	hipMallocManaged(&B, n * sizeof(int));
	hipMallocManaged(&C, n * sizeof(int));

	for (int i = 0; i < n; i++) {
		A[i] = i;
		B[i] = i;
		C[i] = 0;
	}

	vecAdd<<<1,n>>> (A, B, C,n);
	hipDeviceSynchronize();

	for (int i = 0; i < n; i++) {
		printf("C[%d]=%d \n", i, C[i]);

	}
	hipFree(A);
	hipFree(B);
	hipFree(C);

	return 0;



}