﻿
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

void matrixMul(int* a, int* b, int *c, int n, int l) {
	int sum = 0;
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < l; j++) {
			for (int k = 0; k < n; k++) {
				sum += a[i*l+k ] * b[k*l+j ];

			}
			c[i*l+j] = sum;
			sum = 0;
		}
	}



}
int main() {
	int m, n, l, o;
	printf("Enter dimensions of matrix A:\n");
	scanf("%d %d", &m,&n);

	printf("Dimentions of matrix A: %d x %d \n", m, n);

	printf("Enter dimensions of matrix B: \n");
	scanf("%d %d", &l, &o);

	printf("Dimentions of matrix A: %d x %d \n", l, o);

	if (n != l) {
		printf("matrix multiplication is not possible");
		return 0;
	}
	
	int* A, * B,*C ;
	A = (int*)malloc(m * n * sizeof(int));
	B = (int*)malloc(l * o * sizeof(int));
	C = (int*)malloc(n * l * sizeof(int));

	printf("Enter the elements of matrix A: \n");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			printf("A[%d][%d]: ", i, j);
			scanf("%d", &A[i * n + j]);
		}
	}

	printf("Enter the elements of matrix B: \n");
	for (int i = 0; i < l; i++) {
		for (int j = 0; j < o; j++) {
			printf("B[%d][%d]: ", i, j);
			scanf("%d", &B[i * o + j]);
		}
	}


	matrixMul(A, B, C, n, l);

	printf("The resultant matrix is:\n");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < o; j++) {
			printf("%d ", C[i * o + j]);
		}
		printf("\n");
	}
	return 0;
}